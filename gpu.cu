#include "hip/hip_runtime.h"
#include "common.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define NUM_THREADS 1024

int* weights_gpu;
int* visible_bias_gpu;
int* hidden_bias_gpu;

int* intermediate_visible_states;
int* visible_state_gpu;
int* hidden_state_gpu;
hiprandState_t* states;

__global__ void visible_to_hidden_gpu(int* visible_state, int* hidden_state, int* weights, int* visible_bias,
                                      int* hidden_bias, int step, int n_nodes, hiprandState_t* states) {
    int node_idx = blockIdx.x;
    __shared__ int dot_arr[NUM_THREADS];
    dot_arr[threadIdx.x] = 0;

    // NUM_THREADS must be greater than n_nodes!
    if (threadIdx.x < n_nodes) {
        dot_arr[threadIdx.x] = weights[node_idx * n_nodes + threadIdx.x] * visible_state[threadIdx.x];
    }

    __syncthreads();

    // NUM_THREADS must be even!
    // Reduce element products to final dot product
    for (unsigned int s = 1; s < NUM_THREADS; s *= 2) {
        if (threadIdx.x % (2 * s) == 0) {
            dot_arr[threadIdx.x] += dot_arr[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        int dot_product = dot_arr[0] + hidden_bias[node_idx];
        // printf("BlockIdx %d Visible %d%d%d\n", blockIdx.x, visible_state[0], visible_state[1], visible_state[2]);
        // printf("BlockIdx %d V2H Dot Product %d\n\n", blockIdx.x, dot_product);

        double sigmoid = 1. / (1 + exp(-1.0 * dot_product));

        float randunif = hiprand_uniform(&states[node_idx]);
        hidden_state[node_idx] = (sigmoid > randunif) ? 1 : 0;
    }

    return;
}

__global__ void hidden_to_visible_gpu(int* visible_state, int* hidden_state, int* weights, int* visible_bias,
                                      int* hidden_bias, int step, int n_nodes, hiprandState_t* states, bool clamp) {
    int node_idx = blockIdx.x;
    __shared__ int dot_arr[NUM_THREADS];
    dot_arr[threadIdx.x] = 0;

    // NUM_THREADS must be greater than n_nodes!
    if (threadIdx.x < n_nodes) {
        dot_arr[threadIdx.x] = hidden_state[threadIdx.x] * weights[threadIdx.x * n_nodes + node_idx];
    }

    __syncthreads();

    // NUM_THREADS must be even!
    // Reduce element products to final dot product
    for (unsigned int s = 1; s < NUM_THREADS; s *= 2) {
        if (threadIdx.x % (2 * s) == 0) {
            dot_arr[threadIdx.x] += dot_arr[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        int dot_product = dot_arr[0] + visible_bias[node_idx];
        // printf("BlockIdx %d Hidden %d%d%d\n", blockIdx.x, hidden_state[0], hidden_state[1], hidden_state[2]);
        // printf("BlockIdx %d H2V Final Dot Product %d\n\n", blockIdx.x, dot_product);

        double sigmoid = 1. / (1 + exp(-1.0 * dot_product));

        float randunif = hiprand_uniform(&states[node_idx]);
        visible_state[node_idx] = (sigmoid > randunif) ? 1 : 0;

        if (clamp) {
            // Clamping
            if (node_idx == 2) {
                visible_state[node_idx] = 1;
            }
        }
    }

    return;
}

__global__ void init_states(int seed, hiprandState_t* states) {
    // Initialize the state using the provided seed and block id as the sequence number
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

void init_rbm(int n_nodes, int n_weights, int n_steps, int* weights, int* visible_bias, int* hidden_bias, int seed) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here

    hipMalloc((void**)&weights_gpu, n_weights * sizeof(int));
    hipMemcpy(weights_gpu, weights, n_weights * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&visible_bias_gpu, n_nodes * sizeof(int));
    hipMemcpy(visible_bias_gpu, visible_bias, n_nodes * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&hidden_bias_gpu, n_nodes * sizeof(int));
    hipMemcpy(hidden_bias_gpu, hidden_bias, n_nodes * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&intermediate_visible_states, n_steps * sizeof(int));
    hipMalloc((void**)&visible_state_gpu, n_nodes * sizeof(int));
    hipMalloc((void**)&hidden_state_gpu, n_nodes * sizeof(int));

    hipMalloc((void**)&states, n_nodes * sizeof(hiprandState_t));
    init_states<<<n_nodes, NUM_THREADS>>>(seed, states);
}

void reset_rbm(int n_nodes) {
    // Resetting according to Jupyter notebook to all 1s (should be randomized for max cut)
    hipMemset(visible_state_gpu, 0, n_nodes*sizeof(int));
}

void free_rbm() {
    // Free any data objects you may have allocated
    hipFree(weights_gpu);
    hipFree(visible_bias_gpu);
    hipFree(hidden_bias_gpu);
    hipFree(intermediate_visible_states);
    hipFree(visible_state_gpu);
    hipFree(hidden_state_gpu);
    hipFree(states);
}

void simulate_one_step(int n_nodes, int step, int seed, int* trial_visible_vals_gpu, bool clamp) {
    visible_to_hidden_gpu<<<n_nodes, NUM_THREADS>>>(visible_state_gpu, hidden_state_gpu, weights_gpu, 
                                                    visible_bias_gpu, hidden_bias_gpu, step, n_nodes, states);
    hipDeviceSynchronize();
    hidden_to_visible_gpu<<<n_nodes, NUM_THREADS>>>(visible_state_gpu, hidden_state_gpu, weights_gpu, 
                                                    visible_bias_gpu, hidden_bias_gpu, step, n_nodes, states, clamp);
    hipDeviceSynchronize();
    
    // Log current visible node values
    hipMemcpy(trial_visible_vals_gpu + step * n_nodes, visible_state_gpu, n_nodes * sizeof(int), hipMemcpyDeviceToDevice);
}
